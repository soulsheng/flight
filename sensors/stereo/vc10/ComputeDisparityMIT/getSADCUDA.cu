#include "hip/hip_runtime.h"


#include "getSADCUDA.cuh"


	void GetSadCUDA::GetSAD_kernel(uchar* leftImage, uchar* rightImage, uchar* laplacianL, uchar* laplacianR, int nstep, int pxX, int pxY, 
		int blockSize, int disparity, int sobelLimit,
		int x, int y, int blockDim, int *sadArray  )
	{
		// init parameters
		//int blockSize = state.blockSize;
		//int disparity = state.disparity;
		//int sobelLimit = state.sobelLimit;

		// top left corner of the SAD box
		int startX = pxX;
		int startY = pxY;

		// bottom right corner of the SAD box
		int endX = pxX + blockSize - 1;
		int endY = pxY + blockSize - 1;

		//printf("startX = %d, endX = %d, disparity = %d, startY = %d, endY = %d, rows = %d, cols = %d\n", startX, endX, disparity, startY, endY, leftImage.rows, leftImage.cols);

		int leftVal = 0, rightVal = 0;

		int sad = 0;

		for (int i=startY;i<=endY;i++) {
			// get a pointer for this row
			uchar *this_rowL = leftImage + i * nstep;
			uchar *this_rowR = rightImage + i * nstep;

			uchar *this_row_laplacianL = laplacianL + i * nstep;
			uchar *this_row_laplacianR = laplacianR + i * nstep;


				for (int j=startX;j<=endX;j++) {
					// we are now looking at a single pixel value
					/*uchar pxL = leftImage.at<uchar>(i,j);
					uchar pxR = rightImage.at<uchar>(i,j + disparity);

					uchar sL = laplacianL.at<uchar>(i,j);
					uchar sR = laplacianR.at<uchar>(i,j + disparity);
					*/


					uchar sL = this_row_laplacianL[j];//laplacianL.at<uchar>(i,j);
					uchar sR = this_row_laplacianR[j + disparity]; //laplacianR.at<uchar>(i,j + disparity);

					leftVal += sL;
					rightVal += sR;

					uchar pxL = this_rowL[j];
					uchar pxR = this_rowR[j + disparity];

					sad += abs(pxL - pxR);
				}
		}

		//cout << "(" << leftVal << ", " << rightVal << ") vs. (" << leftVal2 << ", " << rightVal2 << ")" << endl;

		int laplacian_value = leftVal + rightVal;

		//cout << "sad with neon: " << sad << " without neon: " << sad2 << endl;


		if (leftVal < sobelLimit || rightVal < sobelLimit)// || diff_score > state.interest_diff_limit)
		{
			sadArray[ y * blockDim + x] =  -1;
		}
		else 
			sadArray[ y * blockDim + x] =  NUMERIC_CONST*(float)sad/(float)laplacian_value;
	}

	void GetSadCUDA::runGetSAD( int row_start, int row_end, int startJ, int stopJ, int * sadArray, uchar* leftImage, uchar* rightImage, uchar* laplacianL, uchar* laplacianR, int nstep, int blockSize, int disparity, int sobelLimit )
	{
#if 1
		int gridY = (row_end - row_start)/blockSize;
		int blockDim = (stopJ - startJ)/blockSize;
		for (int y=0; y< gridY; y++)
		{
			for (int x=0; x< blockDim; x++)
			{
				int i = row_start + y * blockSize;
				int j = startJ + x * blockSize;
				GetSAD_kernel(leftImage, rightImage, laplacianL, laplacianR, nstep, j, i, 
					blockSize, disparity, sobelLimit,
					x, y, blockDim, sadArray );
			}
		}

#else
		for (int i=row_start,iStep = 0; i < row_end; i+=blockSize, iStep++)
		{
			for (int j=startJ, jStep = 0; j < stopJ; j+=blockSize, jStep++)
			{
				// get the sum of absolute differences for this location
				// on both images
				sadArray[ iStep * stopJ + jStep] = GetSAD_kernel(leftImage, rightImage, laplacianL, laplacianR, nstep, j, i, 
					blockSize, disparity, sobelLimit );
			}
		}
#endif
	}
